#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <chrono>
#include <math.h>
#include <string>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "distributions.cu"

// Workaround for intellisense and linter.
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();  // workaround __syncthreads warning
#define KERNEL_ARG2(grid, block)
#define KERNEL_ARG3(grid, block, sh_mem)
#define KERNEL_ARG4(grid, block, sh_mem, stream)
#else
#define KERNEL_ARG2(grid, block) <<< grid, block >>>
#define KERNEL_ARG3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARG4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#endif

// from some other tutorial I found, it is pretty handy.
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void rand_init(const unsigned long long seed, hiprandState *rand_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(seed, id, 0, rand_state + id);
}

__global__ void MHSequence(float *s, const int N, hiprandState *rand_state, float *sigma_step) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = *(rand_state + id);
    float local_sigma_step = *(sigma_step + id);

    float p = hiprand_uniform(&local_state);
    float prob = WS_pdf(p);
    *(s + 0 + id * N) = p;

    for (size_t i = 1; i < N; i++) {
        float dp = local_sigma_step * hiprand_normal(&local_state);
        float nprob = WS_pdf(p + dp);

        if (hiprand_uniform(&local_state) <= nprob / prob) {
            p += dp;
            prob = nprob;
        }

        *(s + i + id * N) = p;
    }
}

// generates a sequence of random numbers using built in gaussian distribution
__global__ void trueGaussian(float *s, const int N, hiprandState *rand_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = *(rand_state + id);

    for (size_t i = 0; i < N; i++) {
        *(s + i + id * N) = hiprand_normal(&local_state);
    }
}

//calculates mean
__host__ float mean(float *s, int a, int b) {
    float sum = 0.0f;
    
    for (size_t i = a; i < b; i++) {
        sum += *(s + i);
    }

    return sum / (b - a);
}

// calculates sample variance
__host__ float s_variance(float *s, int a, int b, float mean) {
    float sum = 0.0f;

    for (size_t i = a; i < b; i++) {
        float tmp = (*(s + i) - mean);
        sum += tmp * tmp;
    }

    return sum / (b - a - 1.0f);
}

// autocorrelation
// https://www.itl.nist.gov/div898/handbook/eda/section3/eda35c.htm
__host__ float auto_corr(float *s, int a, int b, float mean, int lag) {
    float *tmp;
    tmp = (float *) malloc((b - a) * sizeof(float));

    for (size_t i = a; i < b; i++) {
        *(tmp + i - a) = (*(s + i) - mean);
    }

    float sum_a = 0.0f, sum_b = 0.0f;

    for (size_t i = a; i < b - lag; i++) {
        float tmp2 = *(tmp + i - a);
        sum_a += tmp2 * *(tmp + i - a + lag);
        sum_b += tmp2 * tmp2;
    }
    for (size_t i = b - lag; i < b; i++) {
        float tmp2 = *(tmp + i - a);
        sum_b += tmp2 * tmp2;
    }

    free(tmp);

    return (sum_a / sum_b);
}

int main() {
     // Force CUDA context establishment. Not necessary but nvprof output is more understandable.
    checkCudaErrors(hipFree(0));

    const int blocks = 4;
    const int blocksize = 256;
    const int total_threads = blocks * blocksize;
    
    // Initialize random state
    hiprandState *d_rand_state; // d_ prefix is a device pointer
    checkCudaErrors(hipMalloc((void **)&d_rand_state, total_threads * sizeof(hiprandState)));
    const unsigned long long seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    rand_init KERNEL_ARG2(blocks, blocksize) (seed, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());

    // sequence length
    const long long N = 1<<20;

    float *s;
    float *d_s;
    float *sigma_step;
    float *d_sigma_step;

    //treat 1D array like 2D array because it is just easier
    //arr[i][j] becomes arr[j * cols + i]
    s = (float *) malloc(N * total_threads * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&d_s, N * total_threads * sizeof(float)));
    sigma_step = (float *) malloc(total_threads * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&d_sigma_step, total_threads * sizeof(float)));

    const float minSigmaStep = 0.1f;
    const float maxSigmaStep = 5.0f;

    const float dSigmaStep = (maxSigmaStep - minSigmaStep) / total_threads;

    for (size_t i = 0; i < total_threads; i++) {
        *(sigma_step + i) = minSigmaStep + i * dSigmaStep;
    }

    checkCudaErrors(hipMemcpy(d_sigma_step, sigma_step, total_threads * sizeof(float), hipMemcpyHostToDevice));
    
    MHSequence KERNEL_ARG2(blocks, blocksize) (d_s, N, d_rand_state, d_sigma_step);
    //trueGaussian KERNEL_ARG2(blocks, blocksize) (d_s, N, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(s, d_s, N * total_threads * sizeof(float), hipMemcpyDeviceToHost));

    std::string datafile("r1_vs_sigma_step.data");
    std::ofstream data(datafile);

    for (size_t i = 0; i < total_threads; i++) {
        float avg = mean(s, i * N, (i + 1) * N);
        //float var = s_variance(s, i * N, (i + 1) * N, avg);
        float r = auto_corr(s, i * N, (i + 1) * N, avg, 1);
        //std::cout << "Thread " << i << " avg: " << avg << " with var: "<< var <<  " and r: " << r << " where sigma_step: " << *(sigma_step + i) << "\n";
        data << *(sigma_step + i) << ", " << r << "\n";
    }

    data << std::flush;
    data.close();
    
    // free memory
    free(s);
    checkCudaErrors(hipFree(d_s));

    return 0;
}