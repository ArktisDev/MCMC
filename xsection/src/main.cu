#include "hip/hip_runtime.h"
#include "Common.cuh"
#include "Statistics.cuh"
#include "Timing.cuh"
#include "ProgressBar.cuh"
#include "IntegrationOutputHandler.cuh"
#include "Metropolis.cuh"
#include "Integrate.cuh"
#include "Distributions.cuh"

#include <iostream>

// This define for expanding the pdflist in variadic template
#define pdflist pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf, pdf
// In the future it might be useful to include a dummy header for compiling
// where the content of the dummy header is just the define for pdflist.
// That way, an external script can modify the dummy header for the specific
// situation to be compiled, then run the compiler. This makes it so we don't
// have to mess around passing the defines via command line, which is somewhat
// difficult and easy to mess up. 

int main(int argc, char** argv) {
    const int nNeutronA = 6;
    const int nProtonA = 6;
    const int nNeutronB = 6;
    const int nProtonB = 6;
    
    // For 1 simulation kernel
    const int totalBlocks = 1 << 10; // 1024
    const int threadsPerBlock = 1 << 7;    // 128, don't go below this per block (empirical)
    const int samplesPerThread = 1 << 8; // don't go below 1<<9 really (empirical)
    
    const int totalRuns = 1 << 4;  // number of times to iterate the simulation kernel
    
    const int samplesForMix = 1 << 13; // number of times to iterate MH kernel for mixing
    
    // Range of impact parameters to sample
    const float b0 = 0;
    const float db = 0.25;
    // number of db's to step
    const int ndbs = 40;
    
    // Output file
    std::string outDir = "../data/";
    std::string outFile = "S_AB.dat";
    
    // If an arg is supplied, set that to be the output file
    if (argc == 2) {
        outFile = argv[1];
    }
    
    // seed for CUDA RNG
    const uint64_t seed = std::chrono::steady_clock::now().time_since_epoch().count();
    //const uint64_t seed = 8 * 2474033889142906;
    
    
    
    
    
    
    const int nNucleonsA = nNeutronA + nProtonA;
    const int nNucleonsB = nNeutronB + nProtonB;
    const int nNucleons = nNucleonsA + nNucleonsB;
    
    const int64_t totalThreads = totalBlocks * threadsPerBlock;
    const int64_t samplesPerRun = samplesPerThread * totalThreads;
    const int64_t totalSamples = samplesPerRun * totalRuns;
    
    // TODO: maybe log stuff like this in metadata in some file
    std::cout << "Total threads: " << totalThreads << std::endl;
    std::cout << "Samples per thread: " << samplesPerThread << std::endl;
    std::cout << "Total runs: " << totalRuns << std::endl;
    std::cout << "Samples per run: " << samplesPerRun << std::endl;
    std::cout << "Total samples: " << totalSamples << std::endl;
    std::cout << "Total samples log10():" << std::log10(totalSamples) << std::endl;
    
    dim3 blocks(totalBlocks);
    dim3 threads(threadsPerBlock);




    
    // Start of initializing CUDA variables
    
    // A very basic class to do timing
    Timer timer;
    
    // Keep track of random state and data for metropolis hastings
    // Rather than recalculate 1 / pdf(r_prev), just store it between kernel invocations
    hiprandStateXORWOW *d_randState;
    float *d_prevSample;
    
    hipMalloc((void **)&d_randState, totalThreads * sizeof(hiprandStateXORWOW));
    cudaCheckError();
    
    hipMalloc((void **)&d_prevSample, nNucleons * totalThreads * sizeof(float));
    cudaCheckError();
    
    // Init arrays
    RandStateInit<<<blocks, threads>>>(d_randState, seed);
    cudaCheckError();
    InitSampleArray<<<blocks, threads>>>(d_prevSample, 1.0f, nNucleons, totalThreads);
    cudaCheckError();
    
    hipDeviceSynchronize();
    cudaCheckError();
    
    float *d_resultBuffer1;
    float *d_resultBuffer2;
    float *h_resultBuffer1;
    float *h_resultBuffer2;
    
    hipMalloc((void **)&d_resultBuffer1, totalThreads * sizeof(float));
    cudaCheckError();
    hipMalloc((void **)&d_resultBuffer2, totalThreads * sizeof(float));
    cudaCheckError();
    
    h_resultBuffer1 = (float *) malloc(totalThreads * sizeof(float));
    h_resultBuffer2 = (float *) malloc(totalThreads * sizeof(float));
    
    float* d_resultBuffers[2] = {d_resultBuffer1, d_resultBuffer2};
    float* h_resultBuffers[2] = {h_resultBuffer1, h_resultBuffer2};
    
    hipStream_t computeStream, dataStream;
    hipStreamCreate(&computeStream);
    cudaCheckError();
    hipStreamCreate(&dataStream);
    cudaCheckError();
    
    hipEvent_t batches[2];
    for (hipEvent_t& event : batches) {
        hipEventCreateWithFlags(&event, hipEventDisableTiming);
        cudaCheckError();
    }
    
    float runAverages[totalRuns];
    
    // Warmup markov chain
    
    timer.start();
    
    // Start running the chain before taking samples from it, this "mixes" the chain
    // This is an important step, and if we were doing this a little bit better
    // we'd even randomize the initial states of the chain
    // This can be done by treating it as a discrete pdf, and sampling according to that (lars can do that btw lol)
    // Probably it isn't worth it though since we can just mix the chain "enough"
    WarmupMetropolis<nNucleons, totalThreads, pdflist><<<blocks, threads>>>(d_prevSample, 2.2f, d_randState, samplesForMix);
    cudaCheckError();
    hipDeviceSynchronize();
    cudaCheckError();
    
    std::cout << "Warmup took " << timer.elapsedMilli() << "ms" << std::endl;
    
    
    std::vector<std::string> headers = {"ImpactParameter", "S_AB", "Stderr(S_AB)"};
    IntegrationOutputHandler handler(headers.size(), headers, outDir, outFile);
    
    ProgressBar progressBar(ndbs * totalRuns);
    
    std::cout << "Now starting integration" << std::endl;
    
    
    
    
    
    
    
    
    
    
    // Loop over wanted impact parameters
    // dbs = number of db's from b0 to current impact parameter
    for (int dbs = 0; dbs < ndbs; dbs++) {
        float b = b0 + dbs * db;
        // Setup done, now run the integration
        progressBar.IncrementProgress(1);
        progressBar.PrintBar();
        timer.start();
        
        // Launch the first run
        MCIntegrate_S_AB<nNucleonsA, nNucleonsB, totalThreads, pdflist><<<blocks, threads, 0, computeStream>>>(d_prevSample, 2.2, d_randState, d_resultBuffer1, samplesPerThread, b);
        cudaCheckError();
        hipEventRecord(batches[0], computeStream);
        cudaCheckError();
        
        for (int run = 1; run < totalRuns; run++) {
            progressBar.IncrementProgress(1);
            progressBar.PrintBar();
            // Launch a new run
            MCIntegrate_S_AB<nNucleonsA, nNucleonsB, totalThreads, pdflist><<<blocks, threads, 0, computeStream>>>(d_prevSample, 2.2, d_randState, d_resultBuffers[run % 2], samplesPerThread, b);
            cudaCheckError();
            hipEventRecord(batches[run % 2], computeStream);
            cudaCheckError();
            
            // Wait on previous kernel to finish
            hipEventSynchronize(batches[(run + 1) % 2]);
            cudaCheckError();
            
            // Process data from that event
            hipMemcpyAsync(h_resultBuffers[(run - 1) % 2], d_resultBuffers[(run - 1) % 2], totalThreads * sizeof(float), hipMemcpyDeviceToHost, dataStream);
            cudaCheckError();
            hipStreamSynchronize(dataStream);
            cudaCheckError();
            
            runAverages[run - 1] = Average(h_resultBuffers[(run - 1) % 2], totalThreads);
            
            // when done processing, just let loop again and enqueue another kernel, or let exit because we've enqueued enough kernels
        }
        
        // when exiting the loop, there is still one running kernel, with id run = totalRuns - 1.
        hipEventSynchronize(batches[(totalRuns - 1) % 2]);
        cudaCheckError();
        
        // process the data from that event
        hipMemcpyAsync(h_resultBuffers[(totalRuns - 1) % 2], d_resultBuffers[(totalRuns - 1) % 2], totalThreads * sizeof(float), hipMemcpyDeviceToHost, dataStream);
        cudaCheckError();
        hipStreamSynchronize(dataStream);
        cudaCheckError();
        
        runAverages[totalRuns - 1] = Average(h_resultBuffers[(totalRuns - 1) % 2], totalThreads);
        
        int64_t elapsedTime = timer.elapsedMilli();
        
        
        
        
        // Statistics
        
        
        float finalAverage = Average(runAverages, totalRuns);
        float finalVariance = Variance(runAverages, finalAverage, totalRuns);
        float finalStderr = sqrt(finalVariance / totalRuns);
        
        std::vector<float> results = {b, finalAverage, finalStderr};
        handler.AddRow(results);
        
        //std::cout << "Elapsed time: " << elapsedTime / 1000.0 << "s" << std::endl;
        //std::cout << "Integral Samples/s: " << totalSamples / elapsedTime / 1e6 << " GS/s" << std::endl;
    }
    
    progressBar.FinishBar();
    
    handler.WriteToFile();
    
    // Cleanup
    
    hipStreamDestroy(computeStream);
    cudaCheckError();
    hipStreamDestroy(dataStream);
    cudaCheckError();
    
    for (hipEvent_t& event : batches) {
        hipEventDestroy(event);
        cudaCheckError();
    }
    
    hipFree(d_randState);
    cudaCheckError();
    hipFree(d_prevSample);
    cudaCheckError();
    hipFree(d_resultBuffer1);
    cudaCheckError();
    hipFree(d_resultBuffer2);
    cudaCheckError();
    
    free(h_resultBuffer1);
    free(h_resultBuffer2);
    
    return 0;
}