#include <math.h>

#include <hip/hip_runtime.h>


#include <hiprand/hiprand_kernel.h>

// mean = 0.0f, variance = 1.0f, not normalised because it doesn't have to be
__device__ float normal_pdf(float x) {
    return exp(x * x / -2.0f);
}

__device__ float WS_pdf(float x) {
    if (x >= 0) {
        return 1 / (1 + exp((x - 2.0f) / 0.54f));
    } else {
        return 0;
    }
}