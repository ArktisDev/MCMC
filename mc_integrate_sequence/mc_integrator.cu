#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <chrono>
#include <math.h>
#include <string>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>

#include "distributions.cu"

// Workaround for intellisense and linter.
#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();  // workaround __syncthreads warning
#define KERNEL_ARG2(grid, block)
#define KERNEL_ARG3(grid, block, sh_mem)
#define KERNEL_ARG4(grid, block, sh_mem, stream)
#else
#define KERNEL_ARG2(grid, block) <<< grid, block >>>
#define KERNEL_ARG3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARG4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#endif

// from some other tutorial I found, it is pretty handy.
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void rand_init(const unsigned long long seed, hiprandState *rand_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    hiprand_init(seed, id, 0, rand_state + id);
}

__global__ void MHSequence(float *s, const int N, hiprandState *rand_state, const float sigma_step) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = *(rand_state + id);

    float p = hiprand_uniform(&local_state);
    float prob = WS_pdf(p);
    *(s + 0 + id * N) = p;

    for (size_t i = 1; i < N; i++) {
        float dp = sigma_step * hiprand_normal(&local_state);
        float nprob = WS_pdf(p + dp);

        if (hiprand_uniform(&local_state) <= nprob / prob) {
            p += dp;
            prob = nprob;
        }

        *(s + i + id * N) = p;
    }
}

// generates a sequence of random numbers using built in gaussian distribution
__global__ void trueGaussian(float *s, const int N, hiprandState *rand_state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState local_state = *(rand_state + id);

    for (size_t i = 0; i < N; i++) {
        *(s + i + id * N) = hiprand_normal(&local_state);
    }
}

//calculates mean
__host__ float mean(float *s, int a, int b) {
    float sum = 0.0f;
    
    for (size_t i = a; i < b; i++) {
        sum += *(s + i);
    }

    return sum / (b - a);
}


//calculates mean of r^4
__host__ float squared_mean(float *s, int a, int b) {
    float sum = 0.0f;
    
    for (size_t i = a; i < b; i++) {
        float tmp = *(s + i);
        sum += tmp * tmp;
    }

    return sum / (b - a);
}

//calculates mean of r^4
__host__ float quartic_mean(float *s, int a, int b) {
    float sum = 0.0f;
    
    for (size_t i = a; i < b; i++) {
        float tmp = *(s + i);
        sum += tmp * tmp * tmp * tmp;
    }

    return sum / (b - a);
}

// calculates sample variance
__host__ float s_variance(float *s, int a, int b, float mean) {
    float sum = 0.0f;

    for (size_t i = a; i < b; i++) {
        float tmp = (*(s + i) - mean);
        sum += tmp * tmp;
    }

    return sum / (b - a - 1.0f);
}

// autocorrelation
// https://www.itl.nist.gov/div898/handbook/eda/section3/eda35c.htm
__host__ float auto_corr(float *s, int a, int b, float mean, int lag) {
    float *tmp;
    tmp = (float *) malloc((b - a) * sizeof(float));

    for (size_t i = a; i < b; i++) {
        *(tmp + i - a) = (*(s + i) - mean);
    }

    float sum_a = 0.0f, sum_b = 0.0f;

    for (size_t i = a; i < b - lag; i++) {
        float tmp2 = *(tmp + i - a);
        sum_a += tmp2 * *(tmp + i - a + lag);
        sum_b += tmp2 * tmp2;
    }
    for (size_t i = b - lag; i < b; i++) {
        float tmp2 = *(tmp + i - a);
        sum_b += tmp2 * tmp2;
    }

    free(tmp);

    return (sum_a / sum_b);
}

int main() {
     // Force CUDA context establishment. Not necessary but nvprof output is more understandable.
    checkCudaErrors(hipFree(0));

    const int blocks = 32;
    const int blocksize = 256;
    const int total_threads = blocks * blocksize;
    
    // Initialize random state
    hiprandState *d_rand_state; // d_ prefix is a device pointer
    checkCudaErrors(hipMalloc((void **)&d_rand_state, total_threads * sizeof(hiprandState)));
    const unsigned long long seed = std::chrono::high_resolution_clock::now().time_since_epoch().count();
    rand_init KERNEL_ARG2(blocks, blocksize) (seed, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());

    // sequence length
    const long long N = 1<<17;

    float *s;
    float *d_s;

    //treat 1D array like 2D array because it is just easier
    //arr[i][j] becomes arr[j * cols + i]
    s = (float *) malloc(N * total_threads * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&d_s, N * total_threads * sizeof(float)));

    const float sigma_step = 2.0f;
    
    MHSequence KERNEL_ARG2(blocks, blocksize) (d_s, N, d_rand_state, sigma_step);
    //trueGaussian KERNEL_ARG2(blocks, blocksize) (d_s, N, d_rand_state);
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(s, d_s, N * total_threads * sizeof(float), hipMemcpyDeviceToHost));

    //float *m;
    //m = (float *) malloc(total_threads * sizeof(float));

    std::string datafile("mc_integrate.data");
    std::ofstream data(datafile);

    const float PI = 3.14159265358979;

    for (size_t i = 0; i < total_threads; i++) {
        float quartic_avg = quartic_mean(s, i * N, (i + 1) * N);
        float squared_avg = squared_mean(s, i * N, (i + 1) * N);
        //float avg = mean(s, i * N, (i + 1) * N);
        //float var = s_variance(s, i * N, (i + 1) * N, avg);
        //float r = auto_corr(s, i * N, (i + 1) * N, avg, 1);
        float res = (quartic_avg / squared_avg);
        data << res << "\n";
       // *(m + i) = res / 1000;
    }

    data << std::flush;
    data.close();

    //float mean_of_samples = mean(m, 0, N);
   // float variance_of_samples = s_variance(m, 0, N, mean_of_samples);

    //std::cout << "Mean :" << mean_of_samples << " with var = " << variance_of_samples << "\n" << std::flush;
    
    // free memory
    free(s);
    //free(m);
    checkCudaErrors(hipFree(d_s));

    return 0;
}